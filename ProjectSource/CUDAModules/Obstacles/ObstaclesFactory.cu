#include "ObstaclesFactory.cuh"
#include <hip/hip_runtime.h>
#include <cstring>
#include <iostream>

ObstaclesFactory::ObstaclesFactory(size_t width, size_t height) : width_(width), height_(height) {
    hipError_t err = hipMalloc(&obstacle_mask_, width * height * sizeof(bool));
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate obstacle_mask: " + std::string(hipGetErrorString(err)));
    }
    hipMemset(obstacle_mask_, 0, width * height * sizeof(bool));
}

ObstaclesFactory::~ObstaclesFactory() {
    if (obstacle_mask_) {
        hipFree(obstacle_mask_);
    }
}

void ObstaclesFactory::add_circle(float center_x, float center_y, float radius) {
    obstacles_.push_back(std::make_unique<Circle>(center_x, center_y, radius));
    bool* host_mask = new bool[width_ * height_]();
    obstacles_.back()->apply_to_mask(host_mask, width_, height_);
    hipError_t err = hipMemcpy(obstacle_mask_, host_mask, width_ * height_ * sizeof(bool), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        delete[] host_mask;
        throw std::runtime_error("Failed to copy circle mask: " + std::string(hipGetErrorString(err)));
    }
    delete[] host_mask;
}

void ObstaclesFactory::add_rectangle(float center_x, float center_y, float width, float height) {
    obstacles_.push_back(std::make_unique<Rectangle>(center_x, center_y, width, height));
    bool* host_mask = new bool[width_ * height_]();
    obstacles_.back()->apply_to_mask(host_mask, width_, height_);
    hipError_t err = hipMemcpy(obstacle_mask_, host_mask, width_ * height_ * sizeof(bool), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        delete[] host_mask;
        throw std::runtime_error("Failed to copy rectangle mask: " + std::string(hipGetErrorString(err)));
    }
    delete[] host_mask;
}

void ObstaclesFactory::add_custom(const std::vector<std::pair<int, int>>& points) {
    obstacles_.push_back(std::make_unique<Custom>(points));
    bool* host_mask = new bool[width_ * height_]();
    obstacles_.back()->apply_to_mask(host_mask, width_, height_);
    hipError_t err = hipMemcpy(obstacle_mask_, host_mask, width_ * height_ * sizeof(bool), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        delete[] host_mask;
        throw std::runtime_error("Failed to copy custom mask: " + std::string(hipGetErrorString(err)));
    }
    delete[] host_mask;
}

bool* ObstaclesFactory::get_obstacle_mask() const {
    return obstacle_mask_;
}

void ObstaclesFactory::clear_obstacles() {
    obstacles_.clear();
    hipError_t err = hipMemset(obstacle_mask_, 0, width_ * height_ * sizeof(bool));
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to clear obstacle_mask: " + std::string(hipGetErrorString(err)));
    }
}